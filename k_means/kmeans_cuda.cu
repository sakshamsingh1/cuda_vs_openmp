#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include "helper.h"

using namespace std;

#define THREADS_PER_BLOCK 512

/*****************************************************************/

void get_cluster_colors(int cluster_colors[], int, int);
/*** Kernel Definitions ***/
__global__ void clustering(int, int, Point2d*, const Cluster* __restrict__);
__global__ void recenter_sum(int, int, Point2d*, uint4*);
__global__ void convergence_check(Cluster*, uint4*, bool*);
/**** end of the kernel declaration ***/

/*****************************************************************/

int main(int argc, char * argv[]) {
	
	srand((unsigned) time(NULL));
	if(argc != 4 && argc != 6){
		cout<<"usage: kmeans_cuda <K_CLUSTERS> <INPUT_FILE_PATH> <OUTPUT_FILE_PATH>"<<endl;
        exit(1);
    }

	const char *in_file, *out_file;
	int K_clusters = atoi(argv[1]);
	in_file = argv[2]; out_file = argv[3];
	int idx = 0;
	unsigned int img_height, img_width, channels = 3;
	unsigned char* in_image;

	if (!read_png(in_file, &in_image, img_height, img_width, channels)) {
		exit(1);
	}
	int total_img_points = img_height * img_width;
	unsigned char* out_image = (unsigned char*)calloc(total_img_points*3, sizeof(unsigned char*));
	int total_blob_points = get_total_blob_points(in_image, img_height, img_width);
	Point2d* points = (Point2d*)calloc(total_blob_points, sizeof(Point2d));
	int i=0, pt_idx = 0;
	while(i < total_img_points){
		if (in_image[3*i] == 0 && in_image[3*i+1] == 0 && in_image[3*i+2] == 0) {
			points[pt_idx].x = i%img_width;
			points[pt_idx].y = i/img_width;
			points[pt_idx].cluster = -1;
			pt_idx++;
		}
		i++;
	}

	while(idx < total_img_points) {
		out_image[3*idx] = 255;
		out_image[3*idx+1] = 255;
		out_image[3*idx+2] = 255;
		idx++;
	}

	int cluster_colors[K_clusters][3];
	for (int i=0; i<K_clusters; i++) {
		get_cluster_colors(cluster_colors[i], K_clusters, i);
	}

	Cluster* clusters = (Cluster*)calloc(K_clusters, sizeof(Cluster));
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> uniform(0, total_blob_points - 1);
	
	struct timeval tv1, tv2;	
    struct timezone tz;	
    double elapsed;
	i=0;
    gettimeofday(&tv1, &tz);	

	//Initialize clusters by assigning a random 2D Point to the cluster
	while (i<K_clusters) {
		Point2d *point = &points[uniform(gen)];
		clusters[i++] = Cluster(point->x, point->y, 0, (int*)calloc(total_blob_points, sizeof(int)));
	}

    dim3 total_blocks((total_blob_points + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threads_per_block(THREADS_PER_BLOCK);

	//Allocate device memory
	Point2d* d_points;
	Cluster* d_cluster;
	size_t point_size = sizeof(Point2d), cluster_size = sizeof(Cluster);
	bool* d_converged;
	uint4* d_sum;

	hipMalloc((void**)&d_points, total_blob_points * point_size);
	hipMalloc((void**)&d_cluster, K_clusters * cluster_size);
	hipMalloc((void**)&d_converged, sizeof(bool));
	hipMalloc((void**)&d_sum, sizeof(uint4));

	if(!d_points && !d_cluster && !d_sum && !d_converged){
        cout<<"array cannot be allocated"<<endl;
        exit(1);
    }

	hipMemcpy(d_cluster, clusters, K_clusters * cluster_size, hipMemcpyHostToDevice);
	hipMemcpy(d_points, points, total_blob_points * point_size, hipMemcpyHostToDevice);

	bool all_converged = true, converged;
	do {
		clustering<<<total_blocks , threads_per_block>>>(total_blob_points, K_clusters, d_points, d_cluster);
		all_converged = true;
		for (int i = 0; i < K_clusters; i++) {
			hipMemset(d_sum, 0, 4 * sizeof(int));
			recenter_sum<<<total_blocks , threads_per_block>>>(total_blob_points, i, d_points, d_sum);
			convergence_check<<<1, 1>>>(&d_cluster[i], d_sum, d_converged);
			hipMemcpy(&converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost);
			all_converged &= converged;
		}
	} while (!all_converged); // Loop Until Convergence of Centroids

	// copy device memory to host
	hipMemcpy(points, d_points, total_blob_points * point_size, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_cluster, K_clusters * cluster_size, hipMemcpyDeviceToHost);

	// Print the final cluster centre points and the color associated to each cluster
	/*
	for (int i=0; i<K_clusters; i++) {
		cout<<"cluster_rgb_colors: "<<i+1<<cluster_colors[i][0]<<" .. "<<cluster_colors[i][1]<<" .. "<<cluster_colors[i][2]<<endl; ///
		cout<<"final cluster centre-point"<<i+1<<": "<<(int)clusters[i].x<<" "<<(int)clusters[i].y<<endl;	///
	}
	*/

	// free device memory
	hipFree(d_cluster);
	hipFree(d_points);
	hipFree(d_sum);

 	gettimeofday(&tv2, &tz);	
    elapsed = (double) (tv2.tv_sec-tv1.tv_sec) + (double) (tv2.tv_usec-tv1.tv_usec) * 1.e-6;	
    printf("elapsed time = %f seconds.\n", elapsed);

	idx = 0;
	while(idx < total_blob_points){
		int cluster_idx = points[idx].cluster;
		int pos = img_width*points[idx].y + points[idx].x;
		out_image[3*pos] = cluster_colors[cluster_idx][0];
		out_image[3*pos+1] = cluster_colors[cluster_idx][1];
		out_image[3*pos+2] = cluster_colors[cluster_idx][2];
		idx++;
	}

	if (!(write_png(out_file, out_image, img_height, img_width, 3))) {
		cout<<"Failed to write the output .png file"<<endl;
		exit(1);
	}

	delete[] clusters;
	delete[] points;

	return 0;
}

void get_cluster_colors(int cluster_colors[], int K_clusters, int cluster_idx) {
	cluster_idx++;
	int idx = 2;
	if (K_clusters < 7) {
	  	for(int power=4; power>0; power/=2) {
	    	cluster_colors[idx--] = (cluster_idx/power) * 255;
	    	cluster_idx = cluster_idx%power;
		}
	} else {
	  	for(int i=0;i<3;i++) {
	    	cluster_colors[i] = (rand()%K_clusters) * (255/K_clusters);
	  	}
	}
}

__global__ void clustering(int total_img_points, int K_clusters, Point2d* points, const Cluster* __restrict__ clusters) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_img_points){
		Point2d point = points[idx];
		int closest_cluster, dist, min = INT_MAX, i=0;

		while (i < K_clusters){
			dist = square((point.x - clusters[i].x)) + square((point.y - clusters[i].y));
			if (dist < min) {
				min = dist;
				closest_cluster = i;
			}
			i++;
		}
		points[idx].cluster = closest_cluster;
	}
}

__global__ void recenter_sum(int total_img_points, int cluster, Point2d* points, uint4* sumc) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_img_points){
		Point2d point = points[idx];
		if (point.cluster == cluster) {
			atomicAdd(&sumc->x, point.x);
			atomicAdd(&sumc->y, point.y);
			atomicAdd(&sumc->w, 1);
		}
	}

}

__global__ void convergence_check(Cluster* cluster, uint4* sumc, bool* converged) {
	uint32_t total_cluster_points = sumc->w ;
	*converged = false;
	if (total_cluster_points > 0) {
		Cluster copy = *cluster;
		cluster->x = (sumc->x)/total_cluster_points;
		cluster->y = (sumc->y)/total_cluster_points;
		if (cluster->x == copy.x && cluster->y == copy.y)
			*converged=true;
	}
}
