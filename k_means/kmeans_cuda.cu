#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <iostream>	///
#include <cstdlib>
#include "utils.h"

using namespace std;	///
#define square(X) X*X
#define THREADS_PER_BLOCK 512

/*****************************************************************/

/*** Kernel Definitions ***/
__global__ void find_cluster(int, int, Pixel*, const Cluster* __restrict__);
__global__ void recenter1(int, int, Pixel*, uint4*);
__global__ void recenter2(Cluster*, uint4*, bool*);
/**** end of the kernel declaration ***/
void rgb_color_code(int rgb[], int, int);

/*****************************************************************/

int main(int argc, char * argv[]) {
	
	srand((unsigned) time(NULL));
	if(argc != 4){
        fprintf(stderr, "usage: kmeans_sequential <IN_PATH> <OUT_PATH> <K_CLUSTERS> \n");
        exit(1);
    }

	const char *inPath, *outPath;
	inPath = argv[1]; outPath = argv[2];
	int K_clusters = atoi(argv[3]);
	
	unsigned char* image;

	int idx = 0;
	int img_height = 600, img_width = 800;
	int total_pixels = img_height * img_width;
	int total_blobs = 10;
	// int total_blobs = 2 * K_clusters;
	int blob_radius = 150;
	int blob_centres[100][2]; // change
	int pixels_per_blob = 5000;
	int n_pixels = total_blobs * pixels_per_blob;
	Pixel* pixels = (Pixel*)calloc(n_pixels, sizeof(Pixel));
	unsigned char* in_image = (unsigned char*)calloc(total_pixels*3, sizeof(unsigned char*));

	while(idx < total_pixels){
		in_image[3*idx] = 255;
		in_image[3*idx+1] = 255;
		in_image[3*idx+2] = 255;
		idx++;
	}
	int rgb[K_clusters][3];
	for (int i=0; i<K_clusters; i++) {
		rgb_color_code(rgb[i], K_clusters, i);
	}
	
	for (int i=0; i<total_blobs; i++) {
		blob_centres[i][0] = rand()%(img_width - blob_radius); 
		blob_centres[i][1] = rand()%(img_height - blob_radius); 
	}
	int p_idx = 0;
	for (int i=0; i<total_blobs; i++) {
		for (int j=0; j<pixels_per_blob; j++) {
			int x = blob_centres[i][0] + rand()%blob_radius;
			int y = blob_centres[i][1] + rand()%blob_radius;
			int pos = img_width*y + x;
			pixels[p_idx].x = x;
			pixels[p_idx].y = y;
			pixels[p_idx].cluster = -1;
	 		in_image[3*pos] = 0;
			in_image[3*pos+1] = 0;
			in_image[3*pos+2] = 0;
			p_idx++;	
		}
	}

	if ((write_png(inPath, in_image, img_height, img_width, 3)) != 0) {
		cout<<"fail to write input png file"<<endl;
		// printf("fail to write input png file\n");
		exit(1);
	}

	int i=0;

	Cluster* clusters = (Cluster*)calloc(K_clusters, sizeof(Cluster));
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> uniform(0, n_pixels - 1);
	i=0;
	
	//Initialize Cluster and Assign a Random Pixel to the Cluster
	while (i<K_clusters) {
		Pixel *pixel = &pixels[uniform(gen)];
		clusters[i++] = Cluster(pixel->x, pixel->y, 0, 0, (int*)calloc(n_pixels, sizeof(int)));
		cout<<"cluster centres: "<<pixel->x<<" .. "<<pixel->y<<endl; ///
	}
 	cout<<endl;	///

	//Define Blocks and Threads per block
    dim3 numBlocks((n_pixels + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threadsPerBlock(THREADS_PER_BLOCK);

	// allocate device memory
	Pixel* d_pixels;
	Cluster* d_cluster;
	size_t sz_pixel= sizeof(Pixel), sz_cluster = sizeof(Cluster);
	uint4* d_sum;	//To store sum and count
	bool* d_converged;

	hipMalloc((void**)&d_pixels, n_pixels * sz_pixel);
	hipMalloc((void**)&d_cluster, K_clusters * sz_cluster);
	hipMalloc((void**)&d_sum, sizeof(uint4));
	hipMalloc((void**)&d_converged, sizeof(bool));

	if(!d_pixels && !d_cluster && !d_sum && !d_converged){
        printf("cannot allocate array\n");
        exit(1);
    }

	hipMemcpy(d_pixels, pixels, n_pixels * sz_pixel, hipMemcpyHostToDevice);
	hipMemcpy(d_cluster, clusters, K_clusters * sz_cluster, hipMemcpyHostToDevice);


	bool thread_converged = true;
	bool converged;
	int itr = 0;
	do{
		itr++;
		find_cluster<<<numBlocks , threadsPerBlock>>>(n_pixels, K_clusters, d_pixels, d_cluster);
		thread_converged = true;
		for (int i = 0; i < K_clusters; ++i) {
			hipMemset(d_sum, 0, 4 * sizeof(int));

			recenter1<<<numBlocks , threadsPerBlock>>>(n_pixels, i, d_pixels, d_sum);
			recenter2<<<1, 1>>>(&d_cluster[i], d_sum, d_converged);

			hipMemcpy(&converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost);

			thread_converged &= converged;
		}
	} while (!thread_converged || itr > 50000);

	// copy device memory back to host
	hipMemcpy(pixels, d_pixels, n_pixels * sz_pixel, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_cluster, K_clusters * sz_cluster, hipMemcpyDeviceToHost);

	for (int i=0; i<K_clusters; i++) {
		// cout<<"rgb: "<<i<<rgb[i][0]<<" .. "<<rgb[i][1]<<" .. "<<rgb[i][2]<<endl; ///
		cout<<"final cluster "<<i+1<<": "<<(int)clusters[i].x<<" "<<(int)clusters[i].y<<" "<<(int)clusters[i].z<<endl<<endl;	///
	}

	// free device memory
	hipFree(d_pixels);
	hipFree(d_cluster);
	hipFree(d_sum);

	idx = 0;
	unsigned char* out_image = (unsigned char*)calloc(total_pixels*3, sizeof(unsigned char*));
	while(idx < total_pixels){
		out_image[3*idx] = 255;
		out_image[3*idx+1] = 255;
		out_image[3*idx+2] = 255;
		idx++;
	}
	idx = 0;
	while(idx < n_pixels){
		int cluster_idx = pixels[idx].cluster;
		int pos = img_width*pixels[idx].y + pixels[idx].x;
		out_image[3*pos] = rgb[cluster_idx][0];
		out_image[3*pos+1] = rgb[cluster_idx][1];
		out_image[3*pos+2] = rgb[cluster_idx][2];
		idx++;
	}

	if ((write_png(outPath, out_image, img_height, img_width, 3)) != 0) {
		printf("fail to write output png file\n");
		exit(1);
	}

	delete[] clusters;
	delete[] pixels;

	return 0;
}

void rgb_color_code(int rgb[], int K_clusters, int cluster_idx) {
	cluster_idx++;
	int idx = 2;
	if (K_clusters < 7) {
	  	for(int power=4; power>0; power/=2) {
	    	rgb[idx--] = (cluster_idx/power) * 255;
	    	cluster_idx = cluster_idx%power;
		}
	} else {
	  	for(int i=0;i<3;i++) {
	    	rgb[i] = (rand()%K_clusters) * (255/K_clusters);
	  	}
	}
}

__global__ void find_cluster(int total_pixels, int K_clusters, Pixel* pixels, const Cluster* __restrict__ clusters) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_pixels){
		Pixel pixel = pixels[idx];
		int min = INT_MAX, min_cluster, dist, j=0;

		while(j<K_clusters){
			dist = square((pixel.x - clusters[j].x))+ square((pixel.y - clusters[j].y));
			if (dist < min) {
				min = dist;
				min_cluster = j;
			}
			j++;
		}
		pixels[idx].cluster = min_cluster;
	}
}

__global__ void recenter1(int total_pixels, int cluster, Pixel* pixels, uint4* sumc) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_pixels){
		Pixel pixel = pixels[idx];
		if (pixel.cluster == cluster) {
			atomicAdd(&sumc->x, pixel.x);
			atomicAdd(&sumc->y, pixel.y);
			atomicAdd(&sumc->w, 1);
		}
	}

}

__global__ void recenter2(Cluster* cluster, uint4* sumc, bool* converged) {
	uint32_t points = sumc->w ;
	*converged = false;
	if (points > 0) {
		Cluster copy = *cluster;
		cluster->x = (sumc->x) / (points);
		cluster->y = (sumc->y) / (points);
		if (cluster->x == copy.x && 
			cluster->y == copy.y)
			*converged=true;
	}
}
