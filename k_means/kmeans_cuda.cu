#include "hip/hip_runtime.h"
#include <cstdio>
#include <random>
#include <iostream>
#include <cstdlib>
#include <sys/time.h>
#include "helper.h"

using namespace std;

#define THREADS_PER_BLOCK 512

/*****************************************************************/

void get_cluster_colors(int cluster_colors[], int, int);
/*** Kernel Definitions ***/
__global__ void clustering(int, int, Point2d*, const Cluster* __restrict__);
__global__ void recenter_sum(int, int, Point2d*, uint4*);
__global__ void convergence_check(Cluster*, uint4*, bool*);
/**** end of the kernel declaration ***/

/*****************************************************************/

int main(int argc, char * argv[]) {
	
	srand((unsigned) time(NULL));
	if(argc != 4 || argc != 6){
		cout<<"usage: kmeans_cuda <K_CLUSTERS> <INPUT_FILE_PATH> <OUTPUT_FILE_PATH> [<total_blobs> <points_per_blob>]"<<endl;
        exit(1);
    }

	const char *in_file, *out_file;
	int K_clusters = atoi(argv[1]);
	in_file = argv[2]; out_file = argv[3];
	int total_blobs = 10;
	int points_per_blob = 5000;

	// points in input PNG file
	if (argc > 4) {
		int total_blobs = atoi(argv[4]);
		int points_per_blob = atoi(argv[5]);
	}

	int idx = 0, img_height = 600, img_width = 800;
	int total_img_points = img_height * img_width;
	// int total_blobs = 2 * K_clusters;
	int blob_radius = 150;
	int blob_centres[100][2];
	int total_blob_points = total_blobs * points_per_blob;
	Point2d* points = (Point2d*)calloc(total_blob_points, sizeof(Point2d));
	unsigned char* in_image = (unsigned char*)calloc(total_img_points*3, sizeof(unsigned char*));
	unsigned char* out_image = (unsigned char*)calloc(total_img_points*3, sizeof(unsigned char*));

	while(idx < total_img_points){
		in_image[3*idx] = 255;
		in_image[3*idx+1] = 255;
		in_image[3*idx+2] = 255;
		out_image[3*idx] = 255;
		out_image[3*idx+1] = 255;
		out_image[3*idx+2] = 255;
		idx++;
	}

	int cluster_colors[K_clusters][3];
	for (int i=0; i<K_clusters; i++) {
		get_cluster_colors(cluster_colors[i], K_clusters, i);
	}
	
	for (int i=0; i<total_blobs; i++) {
		blob_centres[i][0] = rand()%(img_width - blob_radius); 
		blob_centres[i][1] = rand()%(img_height - blob_radius); 
	}
	int p_idx = 0;
	for (int i=0; i<total_blobs; i++) {
		for (int j=0; j<points_per_blob; j++) {
			int x = blob_centres[i][0] + rand()%blob_radius;
			int y = blob_centres[i][1] + rand()%blob_radius;
			int pos = img_width*y + x;
			points[p_idx].x = x;
			points[p_idx].y = y;
			points[p_idx].cluster = -1;
	 		in_image[3*pos] = 0;
			in_image[3*pos+1] = 0;
			in_image[3*pos+2] = 0;
			p_idx++;	
		}
	}

	if (!(write_png(in_file, in_image, img_height, img_width, 3))) {
		cout<<"Failed to write the input .png file"<<endl;
		exit(1);
	}

	Cluster* clusters = (Cluster*)calloc(K_clusters, sizeof(Cluster));
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_int_distribution<> uniform(0, total_blob_points - 1);
	
	struct timeval tv1, tv2;	
    struct timezone tz;	
    double elapsed;
	int i=0;
    gettimeofday(&tv1, &tz);	

	//Initialize clusters by assigning a random 2D Point to the cluster
	while (i<K_clusters) {
		Point2d *point = &points[uniform(gen)];
		clusters[i++] = Cluster(point->x, point->y, 0, (int*)calloc(total_blob_points, sizeof(int)));
	}

    dim3 total_blocks((total_blob_points + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    dim3 threads_per_block(THREADS_PER_BLOCK);

	//Allocate device memory
	Point2d* d_points;
	Cluster* d_cluster;
	size_t point_size = sizeof(Point2d), cluster_size = sizeof(Cluster);
	bool* d_converged;
	uint4* d_sum;

	hipMalloc((void**)&d_points, total_blob_points * point_size);
	hipMalloc((void**)&d_cluster, K_clusters * cluster_size);
	hipMalloc((void**)&d_converged, sizeof(bool));
	hipMalloc((void**)&d_sum, sizeof(uint4));

	if(!d_points && !d_cluster && !d_sum && !d_converged){
        cout<<"array cannot be allocated"<<endl;
        exit(1);
    }

	hipMemcpy(d_cluster, clusters, K_clusters * cluster_size, hipMemcpyHostToDevice);
	hipMemcpy(d_points, points, total_blob_points * point_size, hipMemcpyHostToDevice);

	bool all_converged = true, converged;
	do {
		clustering<<<total_blocks , threads_per_block>>>(total_blob_points, K_clusters, d_points, d_cluster);
		all_converged = true;
		for (int i = 0; i < K_clusters; i++) {
			hipMemset(d_sum, 0, 4 * sizeof(int));
			recenter_sum<<<total_blocks , threads_per_block>>>(total_blob_points, i, d_points, d_sum);
			convergence_check<<<1, 1>>>(&d_cluster[i], d_sum, d_converged);
			hipMemcpy(&converged, d_converged, sizeof(bool), hipMemcpyDeviceToHost);
			all_converged &= converged;
		}
	} while (!all_converged); // Loop Until Convergence of Centroids

	// copy device memory to host
	hipMemcpy(points, d_points, total_blob_points * point_size, hipMemcpyDeviceToHost);
	hipMemcpy(clusters, d_cluster, K_clusters * cluster_size, hipMemcpyDeviceToHost);

	// Print the final cluster centre points and the color associated to each cluster
	/*
	for (int i=0; i<K_clusters; i++) {
		cout<<"cluster_rgb_colors: "<<i+1<<cluster_colors[i][0]<<" .. "<<cluster_colors[i][1]<<" .. "<<cluster_colors[i][2]<<endl; ///
		cout<<"final cluster centre-point"<<i+1<<": "<<(int)clusters[i].x<<" "<<(int)clusters[i].y<<endl;	///
	}
	*/

	// free device memory
	hipFree(d_cluster);
	hipFree(d_points);
	hipFree(d_sum);

 	gettimeofday(&tv2, &tz);	
    elapsed = (double) (tv2.tv_sec-tv1.tv_sec) + (double) (tv2.tv_usec-tv1.tv_usec) * 1.e-6;	
    printf("elapsed time = %f seconds.\n", elapsed);

	idx = 0;
	while(idx < total_blob_points){
		int cluster_idx = points[idx].cluster;
		int pos = img_width*points[idx].y + points[idx].x;
		out_image[3*pos] = cluster_colors[cluster_idx][0];
		out_image[3*pos+1] = cluster_colors[cluster_idx][1];
		out_image[3*pos+2] = cluster_colors[cluster_idx][2];
		idx++;
	}

	if (!(write_png(out_file, out_image, img_height, img_width, 3))) {
		cout<<"Failed to write the output .png file"<<endl;
		exit(1);
	}

	delete[] clusters;
	delete[] points;

	return 0;
}

void get_cluster_colors(int cluster_colors[], int K_clusters, int cluster_idx) {
	cluster_idx++;
	int idx = 2;
	if (K_clusters < 7) {
	  	for(int power=4; power>0; power/=2) {
	    	cluster_colors[idx--] = (cluster_idx/power) * 255;
	    	cluster_idx = cluster_idx%power;
		}
	} else {
	  	for(int i=0;i<3;i++) {
	    	cluster_colors[i] = (rand()%K_clusters) * (255/K_clusters);
	  	}
	}
}

__global__ void clustering(int total_img_points, int K_clusters, Point2d* points, const Cluster* __restrict__ clusters) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_img_points){
		Point2d point = points[idx];
		int closest_cluster, dist, min = INT_MAX, i=0;

		while (i < K_clusters){
			dist = square((point.x - clusters[i].x))+ square((point.y - clusters[i].y));
			if (dist < min) {
				min = dist;
				closest_cluster = i;
			}
			i++;
		}
		points[idx].cluster = closest_cluster;
	}
}

__global__ void recenter_sum(int total_img_points, int cluster, Point2d* points, uint4* sumc) {
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	if (idx < total_img_points){
		Point2d point = points[idx];
		if (point.cluster == cluster) {
			atomicAdd(&sumc->x, point.x);
			atomicAdd(&sumc->y, point.y);
			atomicAdd(&sumc->w, 1);
		}
	}

}

__global__ void convergence_check(Cluster* cluster, uint4* sumc, bool* converged) {
	uint32_t total_cluster_points = sumc->w ;
	*converged = false;
	if (total_cluster_points > 0) {
		Cluster copy = *cluster;
		cluster->x = (sumc->x)/total_cluster_points;
		cluster->y = (sumc->y)/total_cluster_points;
		if (cluster->x == copy.x && cluster->y == copy.y)
			*converged=true;
	}
}
