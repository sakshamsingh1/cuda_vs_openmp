#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include "utils.h"
#define TILE_DIM 16

using namespace std;

__global__ void mm_kernel(float*, float*, float*, int);

int main(int argc, char** argv){
    int N;
    parseArgs(argc, argv, N);
    size_t allocSize = N*N*sizeof(float);
    
    float* A = (float*)malloc(allocSize);
    float* B = (float*)malloc(allocSize);
    float* C = (float*)malloc(allocSize);

    initializeArray(A, N);
    initializeArray(B, N);

    struct timeval tv1, tv2;
    struct timezone tz;
	double elapsed; 
    
    // Copy array to device
    float* A_c = NULL;
    float* B_c = NULL;
    float* C_c = NULL;
    
	hipMalloc((void**)&A_c, allocSize);
	hipMemcpy(A_c, A, allocSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&B_c, allocSize);
	hipMemcpy(B_c, B, allocSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&C_c, allocSize);


    // Launch kernel
    int bx = ceil(N/(float)TILE_DIM);
    int by = ceil(N/(float)TILE_DIM);
    dim3 dimGrid(bx, by, 1);
    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);

    gettimeofday(&tv1, &tz);
    hipDeviceSynchronize();
    mm_kernel<<<dimGrid, dimBlock>>>(A_c, B_c, C_c, N);
    hipDeviceSynchronize();
    gettimeofday(&tv2, &tz);

    // Copy result back to host
    hipMemcpy(C, C_c, allocSize, hipMemcpyDeviceToHost);

    hipFree(A_c);
    hipFree(B_c);
    hipFree(C_c);

    // printArr(C, N);
/*    bool all_good = checkCorrectness(C, N);
    if(all_good)
        cout << "Correctness check passed" << endl;
    else
        cout << "Correctness check failed" << endl;
 */   
    elapsed = (double) (tv2.tv_sec-tv1.tv_sec) + (double) (tv2.tv_usec-tv1.tv_usec) * 1.e-6;
    printf("elapsed time = %f seconds.\n", elapsed);
}

__global__ void mm_kernel(float* A, float* B, float* C, int N){
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    if(row >= N || col >= N)
        return;
    
    float sum = 0;
    for(int i = 0; i < N; i++){
        sum += A[row*N + i]*B[i*N + col];
    }
    C[row*N + col] = sum;
}
