#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include "utils.h"

#define TILE_DIM 16

using namespace std;

__global__ void mm_kernel_tiled(float*, float*, float*, int);

// matrix multiplication tiling cuda
int main(int argc, char** argv){
    int N;
    parseArgs(argc, argv, N);
    size_t allocSize = N*N*sizeof(float);
    
    float* A = (float*)malloc(allocSize);
    float* B = (float*)malloc(allocSize);
    float* C = (float*)malloc(allocSize);

    initializeArray(A, N);
    initializeArray(B, N);

    struct timeval tv1, tv2;
    struct timezone tz;
	double elapsed; 
    
    // Copy array to device
    float* A_c = NULL;
    float* B_c = NULL;
    float* C_c = NULL;
    
	hipMalloc((void**)&A_c, allocSize);
	hipMemcpy(A_c, A, allocSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&B_c, allocSize);
	hipMemcpy(B_c, B, allocSize, hipMemcpyHostToDevice);

    hipMalloc((void**)&C_c, allocSize);

    // Launch kernel
    int bx = ceil(N/(float)TILE_DIM);
    int by = ceil(N/(float)TILE_DIM);
    dim3 dimGrid(bx, by, 1);
    dim3 dimBlock(TILE_DIM, TILE_DIM, 1);

    gettimeofday(&tv1, &tz);
    hipDeviceSynchronize();
    mm_kernel_tiled<<<dimGrid, dimBlock>>>(A_c, B_c, C_c, N);
    hipDeviceSynchronize();
    gettimeofday(&tv2, &tz);

    // Copy result back to host
    hipMemcpy(C, C_c, allocSize, hipMemcpyDeviceToHost);

    hipFree(A_c);
    hipFree(B_c);
    hipFree(C_c);

    // printArr(C, N);
/*    bool all_good = checkCorrectness(C, N);
    if(all_good)
        cout << "Correctness check passed" << endl;
    else
        cout << "Correctness check failed" << endl;
*/    
    elapsed = (double) (tv2.tv_sec-tv1.tv_sec) + (double) (tv2.tv_usec-tv1.tv_usec) * 1.e-6;
    printf("elapsed time = %f seconds.\n", elapsed);

}

// Kernel
__global__ void mm_kernel_tiled(float* A, float* B, float* C, int N){

    float CValue = 0;

    int Row = blockIdx.y*TILE_DIM + threadIdx.y;
    int Col = blockIdx.x*TILE_DIM + threadIdx.x;

    __shared__ float As[TILE_DIM][TILE_DIM];
    __shared__ float Bs[TILE_DIM][TILE_DIM];

    for (int k = 0; k < (TILE_DIM + N - 1)/TILE_DIM; k++) {

         if (k*TILE_DIM + threadIdx.x < N && Row < N)
             As[threadIdx.y][threadIdx.x] = A[Row*N + k*TILE_DIM + threadIdx.x];
         else
             As[threadIdx.y][threadIdx.x] = 0.0;

         if (k*TILE_DIM + threadIdx.y < N && Col < N)
             Bs[threadIdx.y][threadIdx.x] = B[(k*TILE_DIM + threadIdx.y)*N + Col];
         else
             Bs[threadIdx.y][threadIdx.x] = 0.0;

         __syncthreads();

         for (int n = 0; n < TILE_DIM; ++n)
             CValue += As[threadIdx.y][n] * Bs[n][threadIdx.x];

         __syncthreads();
    }

    if (Row < N && Col < N)
        C[((blockIdx.y * blockDim.y + threadIdx.y)*N) +
           (blockIdx.x * blockDim.x)+ threadIdx.x] = CValue;
}
