#include "hip/hip_runtime.h"
#include <iostream>
#include "util.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

using namespace std;

__global__ void merge_sort_kernel(int*, int*, int, int);

int main(int argc, char** argv){
    int N;
    parseArgs(argc, argv, N);

    // initialize array
    size_t allocSize = N * sizeof(int);
    int *arr = (int*) malloc(allocSize);
    int *tmparr = (int*) malloc(allocSize);

    initializeArray(arr, N);
    
    int *d_arr, *d_tmparr;
    hipMalloc((void**)&d_arr, allocSize);
    hipMalloc((void**)&d_tmparr, allocSize);

    hipMemcpy(d_arr, arr, allocSize, hipMemcpyHostToDevice);
    hipMemcpy(d_tmparr, tmparr, allocSize, hipMemcpyHostToDevice);

    int curr_size;
    for (curr_size=1; curr_size <= N-1; curr_size = 2*curr_size){
        // call kernel
        int K = 1024;
        int num_blocks = ceil((float)N/(float)(K * 2 * curr_size)) + 1;
        int num_threads = K;
        dim3 dimGrid(num_blocks, 1, 1);
        dim3 dimBlock(num_threads, 1, 1);

        merge_sort_kernel<<<dimGrid, dimBlock>>>(d_arr, d_tmparr, curr_size, N);
        hipMemcpy(d_tmparr, d_arr, allocSize, hipMemcpyDeviceToDevice);
    }
    hipMemcpy(arr, d_arr, allocSize, hipMemcpyDeviceToHost);
    hipFree(d_arr);
    hipFree(d_tmparr);

    // bool is_correct = checkCorrectness(arr, N);
    // if (is_correct){
    //     cout << "Correct!" << endl;
    // } else {
    //     cout << "Incorrect!" << endl;
    //     // printArray(arr, N);
    // }

}

__global__ void merge_sort_kernel(int* arr, int* tmp_arr_d, int curr_size, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int left = i * 2 * curr_size;
    if(left > N-1){
        return;
    }
    
    int right = left + curr_size;
    int end = min(right + curr_size - 1, N);

    int left_start = left, right_start = right;
    int left_end = right - 1, right_end = end;
    int result_start = left_start;

    while(left_start < left_end || right_start < right_end){
        if(left_start < left_end && right_start < right_end)
        {
            if(arr[left_start] < arr[right_start]){
                tmp_arr_d[result_start] = arr[left_start];
                left_start += 1;
            }
            else{
                tmp_arr_d[result_start] = arr[right_start];
                right_start += 1;
            }
        }
        else if(left_start < left_end){
            tmp_arr_d[result_start] = arr[left_start];
            left_start += 1;
        }
        else{
            tmp_arr_d[result_start] = arr[right_start];
            right_start += 1;
        }
        
        result_start += 1;
    }

}