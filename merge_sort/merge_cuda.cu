#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include "util.h"

void mergesort(int*, long, dim3, dim3);

__global__ void gpu_mergesort(int*, int*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(int*, int*, long, long, long);

#define min(a, b) (a < b ? a : b)

int main(int argc, char** argv) {

    int size;
    parseArgs(argc, argv, size);

    int *arr = (int*)malloc(size * sizeof(int));
    struct timeval tv1, tv2;
    struct timezone tz;
    double elapsed;

    initializeRandomArray(arr, size);

    dim3 threadsPerBlock(128,32,1);
    dim3 blocksPerGrid(128,32,1);

//    threadsPerBlock.x = 512;
//    threadsPerBlock.y = 512;
//    threadsPerBlock.z = 1;
//
//    blocksPerGrid.x = 512;
//    blocksPerGrid.y = 512;
//    blocksPerGrid.z = 1;

    gettimeofday(&tv1, &tz);
    mergesort(arr, size, threadsPerBlock, blocksPerGrid);
    gettimeofday(&tv2, &tz);
    elapsed = (double) (tv2.tv_sec-tv1.tv_sec) + (double) (tv2.tv_usec-tv1.tv_usec) * 1.e-6;
    printf("elapsed time = %f seconds.\n", elapsed);
}

void mergesort(int* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {
    int* D_data;
    int* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    hipMalloc((void**) &D_data, size * sizeof(int));
    hipMalloc((void**) &D_swp, size * sizeof(int));

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    int* A = D_data;
    int* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
}

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__global__ void gpu_mergesort(int* source, int* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(int* source, int* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}