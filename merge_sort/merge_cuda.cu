#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>
#include "util.h"

long readList(long**);

void mergesort(long*, long, dim3, dim3);

__global__ void gpu_mergesort(long*, long*, long, long, long, dim3*, dim3*);
__device__ void gpu_bottomUpMerge(long*, long*, long, long, long);

int tm();

#define min(a, b) (a < b ? a : b)

int main(int argc, char** argv) {

    int size;
    parseArgs(argc, argv, size);

    int *arr = (int*)malloc(size);
    initializeRandomArray(arr, size);

    dim3 threadsPerBlock();
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    mergesort(arr, size, threadsPerBlock, blocksPerGrid);

    tm();

    // print arr
//    for (int i = 0; i < size; i++) {
//        std::cout << arr[i] << '\n';
//    }
}

void mergesort(long* data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid) {

    long* D_data;
    long* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    // Actually allocate the two arrays
    tm();
    hipMalloc((void**) &D_data, size * sizeof(long));
    hipMalloc((void**) &D_swp, size * sizeof(long));

    // Copy from our input list into the first array
    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);

    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);

    long* A = D_data;
    long* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    tm();
    hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost);
    
    // Free the GPU memory
    hipFree(A);
    hipFree(B);
}

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__global__ void gpu_mergesort(long* source, long* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

__device__ void gpu_bottomUpMerge(long* source, long* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

timeval tStart;
int tm() {
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}