#include "hip/hip_runtime.h"
// sieve of eratosthenes on cuda
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <sys/time.h>
#include "utils.h"

using namespace std;

#define BLOCK_SIZE 1024

__global__ void sieve(bool*, int, int, int, int);

int main(int argc, char** argv){
    int N;
    parseArgs(argc, argv, N);
    bool* is_composite = (bool*)calloc(N+1, sizeof(bool));

    struct timeval tv1, tv2;
    struct timezone tz;
    double elapsed; 

    bool* d_is_composite;
    hipMalloc((void**)&d_is_composite, (N+1)*sizeof(bool));
    hipMemcpy(d_is_composite, is_composite, (N+1)*sizeof(bool), hipMemcpyHostToDevice);

    gettimeofday(&tv1, &tz);
    for(int i = 2; i <= sqrt(N); i++){
        if(!is_composite[i]){
            int start = i*i;
            int end = N;
            int step = i;
            int num_blocks = (end-start)/step/BLOCK_SIZE + 1;
            sieve<<<num_blocks, BLOCK_SIZE>>>(d_is_composite, start, end, step, N);
            hipDeviceSynchronize();
        }
    }
    gettimeofday(&tv2, &tz);
    elapsed = (tv2.tv_sec-tv1.tv_sec) + (tv2.tv_usec-tv1.tv_usec)/1000000.0;
    cout << "Time: " << elapsed << endl;
    hipMemcpy(is_composite, d_is_composite, (N+1)*sizeof(bool), hipMemcpyDeviceToHost);
    hipFree(d_is_composite);
    // printArr(is_composite, N);
    // cout << "Number of primes: " << count << endl;
}

__global__ void sieve(bool* is_composite, int start, int end, int step, int N){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int num = start + idx*step;
    if(num <= N){
        is_composite[num] = true;
    }
}